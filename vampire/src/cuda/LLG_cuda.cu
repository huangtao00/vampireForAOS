
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
//
//  Vampire - A code for atomistic simulation of magnetic materials
//
//  Copyright (C) 2009-2012 R.F.L.Evans
//
//  Email:richard.evans@york.ac.uk
//
//  This program is free software; you can redistribute it and/or modify 
//  it under the terms of the GNU General Public License as published by 
//  the Free Software Foundation; either version 2 of the License, or 
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful, but 
//  WITHOUT ANY WARRANTY; without even the implied warranty of 
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU 
//  General Public License for more details.
//
//  You should have received a copy of the GNU General Public License 
//  along with this program; if not, write to the Free Software Foundation, 
//  Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA.
//
// ----------------------------------------------------------------------------
//
#ifdef CUDA
#include "atoms.hpp"
#include "material.hpp"
#include "errors.hpp"
#include "LLG.hpp"
#include "vcuda.hpp"

//int calculate_spin_fields(const int,const int);
//int calculate_external_fields(const int,const int);

namespace vcuda{
	  bool initf=false;
	  bool initLLGf=false;
	  // device arrays for atoms data


}


namespace vcuda{

int initLLG(){
    return EXIT_SUCCESS;

/*        LLG_arrays::x_spin_storage_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::y_spin_storage_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::z_spin_storage_array.resize(atoms::num_atoms,0.0);

        LLG_arrays::x_initial_spin_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::y_initial_spin_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::z_initial_spin_array.resize(atoms::num_atoms,0.0);

        LLG_arrays::x_euler_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::y_euler_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::z_euler_array.resize(atoms::num_atoms,0.0);

        LLG_arrays::x_heun_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::y_heun_array.resize(atoms::num_atoms,0.0);
        LLG_arrays::z_heun_array.resize(atoms::num_atoms,0.0);

        LLG_arrays::LLG_set=true;*/

}

int LLG(const int num_steps){
    // Function to perform num_steps LLG integration steps on cuda enables device

    // check calling of routine if error checking is activated
    //if(err::check==true){std::cout << "vcuda::LLG has been called" << std::endl;}

    // check cuda atom arrays are initialised
    //if(vcuda::initf==false){
    //  std::cerr << "Warning - cuda device atom arrays are not initialised, initialising" << std::endl; 
    //}
    // check cuda LLG arrays are initialised
    //if(vcuda::initLLGf==false){
    //  std::cerr << "Warning - cuda device LLG arrays are not initialised, initialising" << std::endl;
    //}

    
			



	return EXIT_SUCCESS;
}

} // end of namespace vcuda
#endif

